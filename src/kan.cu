#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <unistd.h>
#include <iomanip>
#include <functional>
#include <memory>
#include <thread>
#include "kan.hpp"
#include "kan_algorithm.hpp"

namespace{
template <class T>
std::unique_ptr<kan_algorithm::kan_base<T>> get_kan_algorithm(const int gpu_id, const int num_sm, const int num_cuda_core_per_sm, kan::algorithm_id algorithm_id){
	kan_algorithm::kan_base<T>* ptr = nullptr;
	switch (algorithm_id) {
	case kan::algorithm_id::gemm:
		ptr = new kan_algorithm::gemm<T>(gpu_id);
	case kan::algorithm_id::julia:
		ptr = new kan_algorithm::julia<T>(gpu_id, num_sm, num_cuda_core_per_sm);
	default:
		; // 世界で一番簡単な文
	}
	return std::unique_ptr<kan_algorithm::kan_base<T>>{ptr};
}
}

template <class T>
void kan::run(const int gpu_id, const int num_sm, const int num_cuda_core_per_sm, kan::algorithm_id algorithm_id, gpu_monitor::string_mode_id string_mode_id){
	// start kan thread {{{
	bool kan_complete = false;
	auto kan_algorithm = get_kan_algorithm<T>(gpu_id, num_sm, num_cuda_core_per_sm, algorithm_id);
	std::thread kan_thread([&kan_algorithm](){kan_algorithm.get()->run(3, {1024});});
	// }}}

	// monitoring GPU {{{
	gpu_monitor::monitor gpu_monitor(gpu_id);
	const auto start_timestamp = std::time(nullptr);
	if(string_mode_id == gpu_monitor::csv){
		std::cerr<<"elapsed_time,";
	}
	std::cerr<<gpu_monitor.get_gpu_status_pre_string(string_mode_id)<<std::endl;
	while(!kan_complete){
		const auto elapsed_time = std::time(nullptr) - start_timestamp;
		if(string_mode_id == gpu_monitor::csv){
			std::cout<<elapsed_time<<",";
		}else{
			std::cout<<"["<<std::setw(6)<<elapsed_time<<"] ";
		}
		std::cout<<gpu_monitor.get_gpu_status_string(string_mode_id)<<std::endl;
		sleep(1);
	}
	// }}}
	kan_thread.join();

	std::cerr<<std::endl;
	std::cerr<<"# Result"<<std::endl
		<<"  - max temperature      : "<<gpu_monitor.get_max_temperature()<<"C"<<std::endl
		<<"  - max power            : "<<(gpu_monitor.get_max_power()/1000.0)<<"W"<<std::endl;
}

template void kan::run<float>(int, int, int, kan::algorithm_id, gpu_monitor::string_mode_id);
template void kan::run<double>(int, int, int, kan::algorithm_id, gpu_monitor::string_mode_id);
// instance
