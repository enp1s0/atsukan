#include <iostream>
#include <exception>
#include <functional>
#include <cxxopts.hpp>
#include <cutf/device.hpp>
#include <hip/hip_runtime_api.h>
#include "kan.hpp"
#include "gpu_monitor.hpp"

namespace{
kan::algorithm_id get_algorithm_id(const std::string algorithm_name){
	if(algorithm_name == "julia") return kan::algorithm_id::julia;
	if(algorithm_name == "gemm") return kan::algorithm_id::gemm;
	throw std::runtime_error("No such an algorithm : " + algorithm_name);
}
gpu_monitor::string_mode_id get_string_mode_id(const std::string string_mode_name){
	if(string_mode_name == "human") return gpu_monitor::string_mode_id::human;
	if(string_mode_name == "csv") return gpu_monitor::string_mode_id::csv;
	throw std::runtime_error("No such printing mode : " + string_mode_name);
}

std::function<void(int, int, int, kan::algorithm_id, gpu_monitor::string_mode_id)> get_run_function(const std::string type_name){
	if(type_name == "float") return [](int g, int a, int b, kan::algorithm_id c, gpu_monitor::string_mode_id d){kan::run<float>(g, a, b, c, d);};
	if(type_name == "double") return [](int g, int a, int b, kan::algorithm_id c, gpu_monitor::string_mode_id d){kan::run<double>(g, a, b, c, d);};
	throw std::runtime_error("No such a type : " + type_name);
}
}

int main(int argc, char** argv){
	const std::string project_name = "High Performance ATSUKAN Computing";
	cxxopts::Options options(project_name, "Options");
	options.add_options()
		("a,algorithm", "Computing algorithm", cxxopts::value<std::string>()->default_value("julia"))
		("g,gpu", "GPU ID", cxxopts::value<unsigned int>()->default_value("0"))
		("p,print_mode", "Printig mdoe", cxxopts::value<std::string>()->default_value("human"))
		("t,type", "Computing type", cxxopts::value<std::string>()->default_value("float"))
		("h,help", "Help");
	const auto args = options.parse(argc, argv);

	// print USAGE {{{
	if(args.count("help")){
		std::cerr<<options.help({""})<<std::endl;
		return 0;
	}
	// }}}
	std::cerr<<project_name<<std::endl;
	std::cerr<<std::endl;

	// print GPU Information {{{
	const auto gpu_id = args["gpu"].as<unsigned int>();

	const auto device_props = cutf::cuda::device::get_properties_vector();
	if(device_props.size() <= gpu_id){
		throw std::runtime_error("No such a GPU : GPU ID = " + std::to_string(gpu_id));
	}
	const auto device_prop = device_props[gpu_id];
	const int num_sm = device_prop.multiProcessorCount;
	const int num_cuda_core_per_sm = _ConvertSMVer2Cores(device_prop.major, device_prop.minor);

	std::cerr
		<<"# Device information"<<std::endl
		<<"  - GPU ID               : "<<gpu_id<<std::endl
		<<"  - GPU name             : "<<device_prop.name<<std::endl
		<<"  - #SM                  : "<<num_sm<<std::endl
		<<"  - #CUDA Cores per a SM : "<<num_cuda_core_per_sm<<std::endl
		<<"  - Clock rate           : "<<device_prop.clockRate<<" kHz"<<std::endl;
	std::cerr<<std::endl;
	// }}}
	

	// print algorithm information {{{
	const auto algorithm_name = args["algorithm"].as<std::string>();
	const auto type_name = args["type"].as<std::string>();
	const auto algorithm_id = get_algorithm_id(algorithm_name);
	const auto run_function = get_run_function(type_name);
	std::cerr
		<<"# Algorithm information"<<std::endl
		<<"  - Algorithm name       : "<<algorithm_name<<std::endl
		<<"  - Computing type       : "<<type_name<<std::endl;
	std::cerr<<std::endl;
	// }}}
	
	// print output information {{{
	const auto string_mode_name = args["print_mode"].as<std::string>();
	const auto string_mode_id = get_string_mode_id(string_mode_name);
	std::cerr
		<<"# Output information"<<std::endl
		<<"  - Output string type   : "<<string_mode_name<<std::endl;
	std::cerr<<std::endl;

	// }
	
	// run {{{
	run_function(gpu_id, num_sm, num_cuda_core_per_sm, algorithm_id, string_mode_id);
	// }}}
}
