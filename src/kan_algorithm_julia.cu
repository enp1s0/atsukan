#include "hip/hip_runtime.h"
#include <type_traits>
#include <cutf/type.hpp>
#include <cutf/memory.hpp>
#include "kan_algorithm.hpp"

namespace{
// convergence_n : 収束とみなす計算回数
template <class T, std::size_t convergence_n = 65536>
__global__ void kernel_julia(T* const output, const std::size_t dim){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= dim * dim) return;

	// 各種定数
	// z = z^z + c
	T z_r = cutf::cuda::type::cast<T>(0.5 - 1.0 * (tid % dim) / dim);
	T z_i = cutf::cuda::type::cast<T>(0.5 - 1.0 * tid / dim);
	T c_r = cutf::cuda::type::cast<T>(-0.3);
	T c_i = cutf::cuda::type::cast<T>(-0.63);

#pragma unroll
	for(auto i = decltype(convergence_n)(0); i < convergence_n; i++){
		z_r = z_r * z_r - z_i * z_i + c_r;
		z_i = cutf::cuda::type::cast<T>(2.0) * z_r * z_i + c_i;
	}
	output[tid] = z_r * z_r + z_i * z_i;
}

}

template <class T>
kan_algorithm::julia<T>::julia(const int gpu_id, const int num_sm, const int num_cuda_core_per_sm) : kan_algorithm::kan_base<T>(gpu_id, num_sm, num_cuda_core_per_sm){
	kan_algorithm::kan_base<T>::arg_ranges.push_back({"dim (field size : dim x dim)", (1<<5), (1<<14), [](const hyperparameter::parameter_t a){return 2 * a;}});
	kan_algorithm::kan_base<T>::arg_ranges.push_back({"threads per a block", (1<<5), (1<<10), [](const hyperparameter::parameter_t a){return 2 * a;}});
}

template <class T>
void kan_algorithm::julia<T>::run(const bool& complete, std::vector<int> parameters){
	const std::size_t dim = parameters[0];
	const std::size_t block_size = parameters[1];

	// memory
	auto d_output = cutf::cuda::memory::get_device_unique_ptr<T>(dim * dim);
	auto h_output = cutf::cuda::memory::get_host_unique_ptr<T>(dim * dim);

	while(!complete){
		kernel_julia<T><<<((dim*dim + block_size - 1)/block_size), block_size>>>(d_output.get(), dim);
		hipDeviceSynchronize();
	}
}

template class kan_algorithm::julia<float>;
template class kan_algorithm::julia<double>;
