#include "kan_algorithm.hpp"

template <class T>
kan_algorithm::gemm<T>::gemm(const int gpu_id) : kan_algorithm::kan_module<T>(gpu_id, 0, 0){}

template <class T>
void kan_algorithm::gemm<T>::run(const int C, std::vector<int>& parameters){
	// 席を計算する行列の大きさ N x N
	std::size_t N = 1 << 12;

	auto dA = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto dB = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto dC = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const T alpha = cutf::cuda::type::cast<T>(0.0f);
	const T beta = cutf::cuda::type::cast<T>(0.0f);

	for(auto c = decltype(C)(0); c < C; c++){
		cutf::cublas::error::check(cutf::cublas::gemm(
				*cublas.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				N, N, N,
				&alpha,
				dA.get(), N,
				dB.get(), N,
				&beta,
				dC.get(), N
				), __FILE__, __LINE__, __func__);
	}
}
