#include <cutf/type.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include "kan_algorithm.hpp"

template <class T>
kan_algorithm::gemm<T>::gemm(const int gpu_id) : kan_algorithm::kan_base<T>(gpu_id, 0, 0){
	kan_algorithm::kan_base<T>::arg_ranges.push_back({"N (matrix size)", (1<<5), (1<<14), [](const hyperparameter::parameter_t a){return 2 * a;}});
}

template <class T>
void kan_algorithm::gemm<T>::run(const bool &complete, std::vector<int> parameters){
	// 席を計算する行列の大きさ N x N
	const std::size_t N = parameters[0];

	auto dA = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto dB = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto dC = cutf::cuda::memory::get_device_unique_ptr<T>(N * N);
	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	const T alpha = cutf::cuda::type::cast<T>(0.0f);
	const T beta = cutf::cuda::type::cast<T>(0.0f);

	while(!complete){
		cutf::cublas::error::check(cutf::cublas::gemm(
				*cublas.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				N, N, N,
				&alpha,
				dA.get(), N,
				dB.get(), N,
				&beta,
				dC.get(), N
				), __FILE__, __LINE__, __func__);
		hipDeviceSynchronize();
	}
}

template class kan_algorithm::gemm<float>;
template class kan_algorithm::gemm<double>;
